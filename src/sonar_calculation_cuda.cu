#include "hip/hip_runtime.h"
// #include <nps_uw_sensors_gazebo/sonar_calculation_cuda.h>

// __global__ 
// void NpsGazeboSonar::CudaTest(float *out, float *a, float *b, int n)
// {
//     for(int i = 0; i < n; i++){
//         out[i] = a[i] + b[i];
//     }
//     printf("CUDA Function running!!");
//     printf("CUDA Function running!!");
//     printf("CUDA Function running!!");
//     printf("CUDA Function running!!");
//     printf("CUDA Function running!!");
//     printf("CUDA Function running!!");
// }

// #include <nps_uw_sensors_gazebo/sonar_calculation_cuda.cuh>

// __device__ double data;

// __global__ void sonar_calculation_kernel(void) {

// 	// insert data to pass
// 	data = 422.146146;
// }

// namespace NpsGazeboSonar {
// 	void sonar_calculation(void)
// 	{
// 		sonar_calculation_kernel <<<1, 1>>> ();

// 		// Pass data
// 		typeof(data) answer;
// 		hipMemcpyFromSymbol(&answer, HIP_SYMBOL(data), sizeof(double), hipMemcpyDeviceToHost);
// 		printf("answer: %f\n", answer);
// 	}
// }

#include <nps_uw_sensors_gazebo/sonar_calculation_cuda.cuh>

#include <math.h>
#include <assert.h>

#define N 100000
#define MAX_ERR 1e-5

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

namespace NpsGazeboSonar {
	void sonar_calculation(void)
	{
		// Check CUDA device
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error!=hipSuccess)
		{
		fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
		exit(-1);
		}

		float *a, *b, *out;
		float *d_a, *d_b, *d_out;

		// Allocate host memory
		a   = (float*)malloc(sizeof(float) * N);
		b   = (float*)malloc(sizeof(float) * N);
		out = (float*)malloc(sizeof(float) * N);

		// Initialize host array
		for(int i = 0; i < N; i++){
			a[i] = 1.0f;
			b[i] = 2.0f;
		}

		// Allocate GPU device memory
		hipMalloc((void**)&d_a, sizeof(float) * N);
		hipMalloc((void**)&d_b, sizeof(float) * N);
		hipMalloc((void**)&d_out, sizeof(float) * N);

		// Copy data from host to GPU Device
		hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	
		// Main function
		vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
		// Transfer data back to host memory
		hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

		// Verification
		for(int i = 0; i < N; i++){
			printf("a = %f\n", a[i]);
			printf("b = %f\n", b[i]);
			printf("out = %f\n", out[i]);
			printf("error = %f\n", fabs(out[i] - a[i] - b[i]));
			assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
		}
		printf("out[0] = %f\n", out[0]);
		printf("PASSED\n");

		// Free GPU memory
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_out);

		// Free host memory
		free(a); 
		free(b); 
		free(out);
	}
}
